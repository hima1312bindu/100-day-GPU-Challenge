#include <iostream>
#include <hip/hip_runtime.h>

#define Mask_width 5  
__constant__ float M[Mask_width];

__global__ void oned_convolution_tiling_kernel(const float* A, float* C, int n) {
    int threadId = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadId;

    __shared__ float S_A[32 + Mask_width - 1];

    if (i < n) {
        S_A[threadId + Mask_width/2] = A[i];
    }

    if (threadId < Mask_width/2) {
        int left_idx = blockIdx.x * blockDim.x - (Mask_width/2) + threadId;
        S_A[threadId] = (left_idx >= 0) ? A[left_idx] : 0.0f;
    }

    if (threadId < Mask_width/2) {
        int right_idx = blockIdx.x * blockDim.x + blockDim.x + threadId;
        S_A[threadId + blockDim.x + Mask_width/2] = (right_idx < n) ? A[right_idx] : 0.0f;
    }

    __syncthreads();

    if (i < n) {
        float result = 0.0f;
        for (int k = 0; k < Mask_width; k++) {
            int idx = threadId + k;
            int global_idx = i + k - Mask_width/2;
            if (global_idx >= 0 && global_idx < n) {
                result += S_A[idx] * M[k];
            }
        }
        C[i] = result;
    }
}

void checkCudaError(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << message << " - CUDA Error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int n = 10;
    float A[n], C[n];
    float d_M[Mask_width];

    for (int i = 0; i < Mask_width; i++) d_M[i] = i;
    for (int i = 0; i < n; i++) A[i] = i;

    float *d_a, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));
    hipMemcpy(d_a, A, n * sizeof(float), hipMemcpyHostToDevice);
    checkCudaError("Failed to copy input data to device");
    hipMemcpyToSymbol(HIP_SYMBOL(M), d_M, Mask_width * sizeof(float));
    checkCudaError("Failed to copy mask data to device");

    dim3 dimBlock(32);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);
    oned_convolution_tiling_kernel<<<dimGrid, dimBlock>>>(d_a, d_c, n);
    checkCudaError("Failed to execute the kernel");

    hipDeviceSynchronize();
    hipMemcpy(C, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    checkCudaError("Failed to copy output data to host");

    hipFree(d_a);
    hipFree(d_c);

    printf("A:\n");
    for (int i = 0; i < n; i++) printf("%.2f ", A[i]);
    printf("\n");

    printf("Mask:\n");
    for (int i = 0; i < Mask_width; i++) printf("%.2f ", d_M[i]);
    printf("\n");

    printf("C:\n");
    for (int i = 0; i < n; i++) printf("%.2f ", C[i]);
    printf("\n");

    return 0;
}
